#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>



/* Pour gérer des images grandes, passer un paramètre nPxPerThreads est chiant, car ça décale
   tout tout le temps. Mais faut le faire...
*/




// shared memory does not work
#define USE_SHARED_MEM 0




/* CUDA is implicitly C++ code, this is needed so out symbols are not mangled */
extern "C" {

#include "dbg.h"
#include "sobel.h"


struct pixel {
        unsigned char R;
        unsigned char G;
        unsigned char B;
        unsigned char A;
};


__constant__ kernel_t kernelX = { {-1, 0, 1}, 
                                  {-2, 0, 2},
                                  {-1, 0, 1} };

__constant__ kernel_t kernelY = { {-1, -2, -1},
                                  { 0,  0,  0},
                                  { 1,  2,  1} };



/* Return the next power of 2 of n, or n if n is already a power of 2 */
static inline uint32_t getNextPowerOf2(uint32_t n) {
    uint32_t cur = n;
    uint32_t rslt = 1;
    /* Find the previous power of 2 */
    while (cur >>= 1) {
        rslt <<= 1;
    }

    if (rslt == n) {
        return n;
    } else {
        return (rslt << 1);
    }
}



__device__ inline unsigned char greyscale_value(struct pixel *pImage, uint32_t pxNum)
{
        /* Get the GreyScale value of our pixel */
        unsigned char R, G, B, greyVal;
        R = pImage[pxNum].R;
        G = pImage[pxNum].G;
        B = pImage[pxNum].B;

        greyVal = (R + G + B) / 3;

        return greyVal;
}




/* pxNum is the index of the central pixel for the convolution in the pImage array */
__device__ inline int32_t convolution_by_3(struct pixel *pImage, kernel_t kernel,
                                           uint32_t pxNum, uint32_t width, uint32_t height)
{
        int32_t grad = 0;
        /* Line below */
        grad += kernel[0][0] * greyscale_value(pImage, pxNum + width + 1);
        grad += kernel[0][1] * greyscale_value(pImage, pxNum + width);
        grad += kernel[0][2] * greyscale_value(pImage, pxNum + width - 1);

        /* current line */
        grad += kernel[1][0] * greyscale_value(pImage, pxNum + 1);
        grad += kernel[1][1] * greyscale_value(pImage, pxNum);
        grad += kernel[1][2] * greyscale_value(pImage, pxNum - 1);

        /* line above */
        grad += kernel[2][0] * greyscale_value(pImage, pxNum - width + 1);
        grad += kernel[2][1] * greyscale_value(pImage, pxNum - width);
        grad += kernel[2][2] * greyscale_value(pImage, pxNum - width - 1);

        return grad;
}




#if 0
__global__ void sobel_unnorm_kernel(struct pixel *pInImageData, uint16_t *pOutImageData,
                                    uint32_t width, uint32_t height, int numWorkerThreads)
{
    return;
}
    
#else


__global__ void sobel_unnorm_kernel(struct pixel *pInImageData, uint16_t *pOutImageData,
                                    uint32_t width, uint32_t height, int basePx)
{
    /* Ignore basePx for the moment */
    (void) basePx;
    uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x;

    /* Copy all pixels we are responsible for. The first one is our position in the grid. */

    //XXX: we could be better by doing things more "locally"... But harder
    /*for (uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x; [> First pixel is my position in the<]*/
                  /*pxNum < width * height; pxNum += numWorkerThreads) {*/

        /* If we are on a border, do nothing */
        if (   pxNum < width /* First line */
                || pxNum % width == 0 /* First column */
                || pxNum % width == width - 1 /* last column */
                || pxNum >= (width * (height - 1)) /* Last line */
           )
        {
            pOutImageData[pxNum] = 0;
        }
        else
        {
            int32_t gradX = convolution_by_3(pInImageData, kernelX, pxNum, width, height);
            int32_t gradY = convolution_by_3(pInImageData, kernelY, pxNum, width, height);
            float gradX_float = (float) gradX;
            float gradY_float = (float) gradY;

            uint16_t normGrad = (uint32_t) sqrt(gradX_float*gradX_float + gradY_float*gradY_float);

            pOutImageData[pxNum] = normGrad;
        }

    /*}*/
}
#endif



/* This kernel will only handle the normalization */
__global__ void norm_image_kernel(uint16_t *pMaxGrads, uint16_t *pNonNormalized, struct pixel *pOutImage,
                                  uint32_t width, uint32_t height, int basePx)
{
    /* Ignore basePx for the moment */
    (void) basePx;
    uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x;

    uint16_t maxGrad = pMaxGrads[0];

    //XXX divergent kernels, not super efficient
    /*for (uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x; [> First pixel is my position in the<]*/
                  /*pxNum < width * height; pxNum += numWorkerThreads) {*/

        /* If we are on a border, do nothing */
        if (   pxNum < width /* First line */
                || pxNum % width == 0 /* First column */
                || pxNum % width == width - 1 /* last column */
                || pxNum >= (width * (height - 1)) /* Last line */
           )
        {
            //XXX Que faire ici ?
            pOutImage[pxNum].R = 0;
            pOutImage[pxNum].G = 0;
            pOutImage[pxNum].B = 0;
            pOutImage[pxNum].A = 255; /* Full opacity */
        }
        else
        {
            unsigned char greyVal = (255 * pNonNormalized[pxNum]) / maxGrad;
            pOutImage[pxNum].R = greyVal;
            pOutImage[pxNum].G = greyVal;
            pOutImage[pxNum].B = greyVal;
            pOutImage[pxNum].A = 255; /* Full transparency */
        }

    /*}*/
}



__global__ void max_reduction_kernel(uint16_t *pMaxGrads, uint32_t width, uint32_t height)
{
    extern __shared__ uint16_t sData[]; /* Contains the data for the local reduction */

    uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t tid = threadIdx.x;

    /* For each of the pixels the thread is responsible for */
    //XXX il est possible que ceci marche... mais chaud à l'invocation !
    /*for (uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x; [> First pixel is my position in the<]*/
                  /*pxNum < width * height; pxNum += numWorkerThreads) {*/

        /* Each thread copies its pixel */
    if (pxNum < width * height) {
        sData[tid] = pMaxGrads[pxNum];
    } else {
        sData[tid] = 0;
    }

    __syncthreads();

        /* Now, reduce in parallel to find the max */
        for (uint32_t stride = blockDim.x / 2; stride > 0; stride = stride >> 1) {
            if (tid < stride) { /* If we are on the lowest part of the remaining array */
                sData[tid] = max(sData[tid], sData[tid + stride]);
            }

            __syncthreads();
        }

        if (tid == 0) {
            pMaxGrads[blockIdx.x] = sData[0];
        }

        /*if (tid == 0) {*/
            /*pMaxGrads[blockIdx.x] = max(pMaxGrads[blockIdx.x], sData[0]);*/
        /*}*/
    /*}*/

    /* At the end of that kernel, pOutData contains the max value of each block */
}





void log_time(FILE *logFile, char *testName, uint32_t size, double t, int numThreads)
{
        //XXX should do something
}



int sobel(struct image *const pInImage, struct image *pOutImage)
{
        check_warn (pInImage->type == RGBA, "In image must be RGBA");

        uint32_t width = pInImage->width;
        uint32_t height = pInImage->height;

        hipError_t ret;
        struct hipDeviceProp_t deviceProp;

        // We will only use one device, the first one
        ret = hipGetDeviceProperties(&deviceProp, 0);

        int maxThreadsPerBlock = deviceProp.maxThreadsDim[0];
        int nbPx = width * height;

        /* Number of running threads per block, take care of the case
           where there are very little pixels */
        int nThreadsPerBlock = nbPx > maxThreadsPerBlock ? maxThreadsPerBlock : nbPx;

        /* And number of blocks to cover all pixels */
        int nBlocks = nbPx / nThreadsPerBlock + (nbPx % nThreadsPerBlock == 0 ? 0 : 1);
        /*int maxConcurrentBlocks = deviceProp.maxGridSize[0];*/
        /*int maxConcurrentThreads = maxConcurrentBlocks * nThreadsPerBlock;*/
        


        //XXX toujours faire attention au cas où pas assez de threads pour
        //XXX tous les pixels... On ignore pour l'instant, on pourra passer
        //XXX un baseIndex aux kernels, probablement.
        /*if (gridLength > deviceProp.maxGridSize[0]) {*/


        /* Allocate memory on the device for in image, and non-normalized gradient norms. */
        struct pixel *inImageDevice = NULL;
        uint16_t *outNonNormalized = NULL;

        ret = hipMalloc((void **)&inImageDevice, width * height * sizeof(struct pixel));
        check_warn (ret == hipSuccess, "Failed to allocate memory for in image on the device");

        ret = hipMalloc((void **)&outNonNormalized, width * height * sizeof(uint16_t));
        check_warn (ret == hipSuccess, "Failed to allocate memory for out image on the device");

        /* Copy the input image to the device */
        ret = hipMemcpy(inImageDevice, pInImage->data, width * height * sizeof(struct pixel),
                         hipMemcpyHostToDevice);
        check_warn (ret == hipSuccess, "Failed to copy input image to device");


        /* Now, we need to invoke the sobel kernel, that will make convolutions.
           We must be careful to invoke it as much times as necessary considering
           that there might be more pixels than allocatable threads. */
        //XXX later
        /*for (int basePx = 0; basePx < nbPx; basePx += maxConcurrentThreads) {*/
            /* Number of blocks needed for this invokation */
            /*int curNBlocks = (nbPx - basePx > maxConcurrentThreads)*/
                                    /*? maxConcurrentBlocks*/
                                    /*: 1 + (nbPx - basePx) / nThreadsPerBlock;*/

            int curNBlocks = nBlocks;
                                    
            /* No local memory for this kernel, although it could benefit it.
               XXX see later ! */
            /* dummy */ int basePx = 0;
            sobel_unnorm_kernel <<< curNBlocks, nThreadsPerBlock >>> (inImageDevice, outNonNormalized, width, height, basePx);
        /*}*/



        /* Now, we need to get the maximum gradient norm value located in outNonNormalized.
           We don't need the input image any longer */
        hipFree(inImageDevice);


        /* Allocate memory for max grad reduction, and copy non-normalized data */
        uint16_t *maxGrads = NULL;

        ret = hipMalloc((void **)&maxGrads, width * height * sizeof(uint16_t));
        check_warn (ret == hipSuccess, "Failed to allocate memory for out image on the device");
        ret = hipMemcpy(maxGrads, outNonNormalized, width * height * sizeof(uint16_t),
                         hipMemcpyDeviceToDevice);

        /* And invoke iteratively out max-reduction kernel */
        /* Each pass will reduce the number of elements by a factor of nThreadsPerBlock */
        //XXX Sûr ?
        //XXX Faire encore attention au nombre de threads trop petit pour le nombre de pixels...
        //XXX PLUS TARD ! Pour ça, à cet endroit-là, juste gérer ça au moment du load
        //XXX depuis la mémoire globale: gérer 10 px chacun si nécessaire.
#if 0
        int nPasses = 1 + nBlocks / maxThreadsPerBlock;
        int curBlocks = nBlocks;
        for (int i = 0; i < nPasses; i++) {
            /* Number of threads per block for this iteration,  */
            int tpb = (curBlocks > maxThreadsPerBlock) ? maxThreadsPerBlock : curBlocks;
            curBlocks = curBlocks / tpb; //XXX Pourquoi faire ça ici et pas APRES l'invocation du kernel ?
            /* Shared memory per block: one int16 for each thread */
            size_t sharedMemSize = tpb * sizeof(int16_t);
            max_reduction_kernel <<< curBlocks, tpb, sharedMemSize >>> (maxGrads, width, height);


            int16_t maxGrad;
            hipMemcpy(&maxGrad, maxGrads, sizeof(int16_t), hipMemcpyDeviceToHost);
            printf("After %d iteration: en tête du tableau: %d\n", i, maxGrad);

            /* After this iteration, we have reduced each block of tpb values into one
               maximum. We can now call that same kernel with tpb times less threads */
            /*curBlocks = curBlocks / tpb; //XXX Pourquoi faire ça ici et pas APRES l'invocation du kernel ?*/
        }
#else
        //XXX For debug ppurposes: fihd the real max */
        {
            uint16_t *maxes = (uint16_t *)calloc(width * height, sizeof(uint16_t));
            hipMemcpy(maxes, maxGrads, width * height * sizeof(uint16_t), hipMemcpyDeviceToHost);

            uint16_t theMax = 0;
            for (int i = 0; i < width * height; i++) {
                if (maxes[i] > theMax) {
                    theMax = maxes[i];
                }
            }
            printf("The real max is: %u\n", theMax);
        }

        int i = 0;//XXX for debug
        uint32_t remainingElems = nbPx;
        while (remainingElems > 1) {
            uint32_t threadsPerBlock = min(remainingElems, maxThreadsPerBlock);
            /* Be careful, the kernel only works if the number of threads per block is
               a power of 2 */
            threadsPerBlock = getNextPowerOf2(threadsPerBlock);
            uint32_t nBlocks = remainingElems / threadsPerBlock + (remainingElems % threadsPerBlock == 0 ? 0 : 1);
            uint32_t sharedMem = threadsPerBlock * sizeof(uint16_t);


            printf("Before invocation n° %d, nBlocks = %u, threadsPerBlock = %u, sharedMem = %u,"
                    "remainingElems = %u\n", i, nBlocks, threadsPerBlock, sharedMem, remainingElems);

            max_reduction_kernel <<< nBlocks, threadsPerBlock, sharedMem >>> (maxGrads, width, height);

            remainingElems = remainingElems / threadsPerBlock + (remainingElems % threadsPerBlock == 0 ? 0 : 1);
            i++;
        }
            
#endif

        /* For debug, get max */
        uint16_t maxGrad;
        hipMemcpy(&maxGrad, maxGrads, sizeof(int16_t), hipMemcpyDeviceToHost);
        printf("Max grad : %u\n", maxGrad);


        //XXX faire un truc pour les bords aussi... padder avec des zéros DANS LA MÉMOIRE ALLOUÉE SUR LE DEVICE

        /* Allocate memory for the final resulting image */
        struct pixel *outNormalizedDevice = NULL;
        ret = hipMalloc((void **) &outNormalizedDevice, width * height * sizeof(struct pixel));
        check_warn(ret == hipSuccess, "Failed to allocate memory for outNormalizedDevice");

        /* Now, it's time to call the kernel that normalises the image gradients and puts it
           into pixels */
        curNBlocks = nBlocks;

        /* No local memory for this kernel, although it could benefit it.
           XXX see later ! */
        norm_image_kernel <<< curNBlocks, nThreadsPerBlock >>>
            (maxGrads, outNonNormalized, outNormalizedDevice, width, height, basePx);

        
        /* Copy the result from device, and we're done */

        pOutImage->width = width;
        pOutImage->height = height;
        pOutImage->type = RGBA;
        pOutImage->data = (unsigned char*)calloc(pOutImage->width * pOutImage->height, sizeof(struct pixel));

        hipMemcpy(pOutImage->data, outNormalizedDevice, width * height * sizeof(struct pixel),
                   hipMemcpyDeviceToHost);
        return 0;



        //XXX ancien code
#if 0

        ret = hipMalloc((void **)&outImageDevice, width * height * sizeof(struct pixel));
        check_warn (ret == hipSuccess, "Failed to allocate memory for out image on the device");

        /* We will also need memory for the max-reduction, and the non-normalized gradient norms */





        dim3 threadsPerBlock(maxLinearThreads);

        int gridLength = (width * height) / maxLinearThreads +
                        ((width * height) % maxLinearThreads == 0 ? 0 : 1);

        /* If that's too much blocks, reduce, and each thread will handle several pixels
           (handled in the kernel) */
        if (gridLength > deviceProp.maxGridSize[0]) {
            gridLength = deviceProp.maxGridSize[0];
        }

        uint32_t numWorkerThreads = gridLength * maxLinearThreads;

        dim3 nBlocks(gridLength);


        printf("%d blocks of %d threads each, for %d total worker threads, and %d pixels\n",
                gridLength, maxLinearThreads, gridLength * maxLinearThreads, width*height);


        struct pixel *inImageDevice;
        uint16_t *outImageDevice;
        struct pixel *outNormalizedDevice;

        /* Allocate memory on the device for both images (in and out) */
        ret = hipMalloc((void **)&inImageDevice, width * height * sizeof(struct pixel));
        check_warn (ret == hipSuccess, "Failed to allocate memory for in image on the device");

        ret = hipMalloc((void **)&outImageDevice, width * height * 4 * sizeof(uint16_t));
        check_warn (ret == hipSuccess, "Failed to allocate memory for out image on the device");

        ret = hipMalloc((void **)&outNormalizedDevice, width * height * sizeof(struct pixel));
        check_warn (ret == hipSuccess, "Failed to allocate memory for out normalized, image on the device");

        /* Copy The input image on the device */
        ret = hipMemcpy(inImageDevice, pInImage->data, width * height * sizeof(struct pixel),
                         hipMemcpyHostToDevice);
        check_warn (ret == hipSuccess, "Failed to copy the image to the device");


        /* Unnormalized version */
        uint16_t *pUnNormalizedOut = NULL;
        pUnNormalizedOut = (uint16_t *)calloc(width*height, sizeof(uint16_t));

        /* Normalized output */
        //XXX
        struct pixel *pNormalizedOut = NULL;
        pNormalizedOut = (struct pixel *)calloc(width*height, sizeof(struct pixel));

        /* Allocate memory for the resulting image */
        pOutImage->width = pInImage->width;
        pOutImage->height = pInImage->height;
        pOutImage->type = RGBA;
        pOutImage->data = (unsigned char*)calloc(pOutImage->width * pOutImage->height, sizeof(struct pixel));
        /*check_mem(pOutImage->data);*/

        /* And launch the kernel */
        sobel_unnorm_kernel <<< nBlocks, threadsPerBlock >>> (inImageDevice, outImageDevice, width, height, numWorkerThreads);

        ret = hipMemcpy(pUnNormalizedOut, outImageDevice,
                         width * height * sizeof(uint16_t), hipMemcpyDeviceToHost);
        check_warn (ret == hipSuccess, "Kernel failed: %s", hipGetErrorString(ret));


        /* Normalize, on CPU for the moment */
        uint16_t maxGrad = 0;
        //XXX maxPos for debug only
        uint32_t maxPos = 0;
        for (uint32_t i = 0; i < width*height; i++) {
                if (pUnNormalizedOut[i] > maxGrad) {
                        maxGrad = pUnNormalizedOut[i];
                        maxPos = i;
                }
        }
        printf("Max: %u, at %u\n", maxGrad, maxPos);

#if 1
        //XXX this should NOT be necessary
        ret = hipMemcpy(outImageDevice, pUnNormalizedOut,
                         width * height * sizeof(uint16_t), hipMemcpyHostToDevice);

        /* Normalization kernel */
        norm_image_kernel <<< nBlocks, threadsPerBlock >>> (outImageDevice, outNormalizedDevice,
                                                            maxGrad, width, height, numWorkerThreads);


        hipMemcpy(pOutImage->data, (unsigned char *) outNormalizedDevice,
                   width*height*sizeof(struct pixel), hipMemcpyDeviceToHost);

        for (int i = 0; i < 20; i++) {
            printf("%d ", pOutImage->data[i]);
        }
#else

        for (uint32_t i = 0; i < width*height; i++) {
                unsigned char greyVal = (255 * pUnNormalizedOut[i]) / maxGrad;
                pOutImage->data[4*i] = greyVal;
                pOutImage->data[4*i + 1] = greyVal;
                pOutImage->data[4*i + 2] = greyVal;
                pOutImage->data[4*i + 3] = 255; /* full opacity */
        }
#endif




        hipFree(inImageDevice);
        hipFree(outImageDevice);
        hipFree(outNormalizedDevice);
        return 0;
        //XXX need a cleanup for image in case of failure.

/*error:*/
        /*hipFree(inImageDevice);*/
        /*hipFree(outImageDevice);*/
        /*free_and_null(pOutImage->data);*/
        /*return -1;*/
#endif /* old code */
}




} /* extern "C" */
