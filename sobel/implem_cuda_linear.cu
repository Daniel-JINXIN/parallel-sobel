#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>



/* Pour gérer des images grandes, passer un paramètre nPxPerThreads est chiant, car ça décale
   tout tout le temps. Mais faut le faire...
*/




// shared memory does not work
#define USE_SHARED_MEM 0




/* CUDA is implicitly C++ code, this is needed so out symbols are not mangled */
extern "C" {

#include "dbg.h"
#include "sobel.h"


/*int maxNumThreads(int *pMaxGridWidth, int *pMaxGridHeight, int *pMaxBlockDim);*/
/*static inline int maxSquareDim(int numItems);*/


//XXX should change the world to use this...
struct pixel {
        unsigned char R;
        unsigned char G;
        unsigned char B;
        unsigned char A;
};


__constant__ kernel_t kernelX = { {-1, 0, 1}, 
                                  {-2, 0, 2},
                                  {-1, 0, 1} };

__constant__ kernel_t kernelY = { {-1, -2, -1},
                                  { 0,  0,  0},
                                  { 1,  2,  1} };


__device__ inline unsigned char greyscale_value(struct pixel *pImage, uint32_t pxNum)
{
        /* Get the GreyScale value of our pixel */
        unsigned char R, G, B, greyVal;
        R = pImage[pxNum].R;
        G = pImage[pxNum].G;
        B = pImage[pxNum].B;

        greyVal = (R + G + B) / 3;

        return greyVal;
}




//XXX We do redundant computation between threads, can be improved with shared memory
/* pxNum is the index of the central pixel for the convolution in the pImage array */
__device__ inline int32_t convolution_by_3(struct pixel *pImage, kernel_t kernel,
                                           uint32_t pxNum, uint32_t width, uint32_t height)
{
        int32_t grad = 0;
        /* Line below */
        grad += kernel[0][0] * greyscale_value(pImage, pxNum + width + 1);
        grad += kernel[0][1] * greyscale_value(pImage, pxNum + width);
        grad += kernel[0][2] * greyscale_value(pImage, pxNum + width - 1);

        /* current line */
        grad += kernel[1][0] * greyscale_value(pImage, pxNum + 1);
        grad += kernel[1][1] * greyscale_value(pImage, pxNum);
        grad += kernel[1][2] * greyscale_value(pImage, pxNum - 1);

        /* line above */
        grad += kernel[2][0] * greyscale_value(pImage, pxNum - width + 1);
        grad += kernel[2][1] * greyscale_value(pImage, pxNum - width);
        grad += kernel[2][2] * greyscale_value(pImage, pxNum - width - 1);

        return grad;
}




__global__ void sobel_unnorm_kernel(struct pixel *pInImageData, uint16_t *pOutImageData,
                                    uint32_t width, uint32_t height)
{
    /* Get your pixel number, which is your global threadIdx */
    uint32_t pxNum = blockIdx.x * blockDim.x + threadIdx.x;

    /* Be careful with borders */
    if (pxNum > width * height) {
        return;
    }

    /* If we are on a border, do nothing */
    if (   pxNum < width /* First line */
        || pxNum % width == 0 /* First column */
        || pxNum % width == width - 1 /* last column */
        || pxNum >= (width * (height - 1)) /* Last line */
       )
    {
        pOutImageData[pxNum] = 0; // We could also wait and get the max
    }
    else
    {
        int32_t gradX = convolution_by_3(pInImageData, kernelX, pxNum, width, height);
        int32_t gradY = convolution_by_3(pInImageData, kernelY, pxNum, width, height);
        float gradX_float = (float) gradX;
        float gradY_float = (float) gradY;

        uint16_t gradNorm = (uint32_t) sqrt(gradX_float*gradX_float + gradY_float*gradY_float);

        pOutImageData[pxNum] = gradNorm;
    }
    
    //XXX there should be the normalization on the device here !
}



void log_time(FILE *logFile, char *testName, uint32_t size, double t, int numThreads)
{
        //XXX should do something
}



int sobel(struct image *const pInImage, struct image *pOutImage)
{
        check_warn (pInImage->type == RGBA, "In image must be RGBA");

        uint32_t width = pInImage->width;
        uint32_t height = pInImage->height;

        hipError_t ret;
        struct hipDeviceProp_t deviceProp;

        // We will only use one device, the first one
        ret = hipGetDeviceProperties(&deviceProp, 0);
        //XXX check ret

        /*int maxBlocks = deviceProp.maxGridSize[0];*/
        //XXX on pourrait juste autoriser les blocs à être carrés, mais la grille serait linéaire
        int maxLinearThreads = deviceProp.maxThreadsDim[0];

        dim3 threadsPerBlock(maxLinearThreads);

        int gridLength = (width * height) / maxLinearThreads +
                        ((width * height) % maxLinearThreads == 0 ? 0 : 1);

        dim3 nBlocks(gridLength);

        //XXX handle the case where there are not enough...
        /*if (gridWidth * gridHeight > maxGridWidth) { [> maxGridWidth is also the max number of blocks (I think) <]*/
            /* We will have to cut... But how do we do this ?? */
        /*}*/
        /* The number of invoked worker threads, used by kernels to know if they have
           to treat several pixels */
        /*int numThreads = gridWidth * gridHeight * maxBlockDim * maxBlockDim;*/


        printf("%d blocks of %d threads each, for %d total worker threads, and %d pixels\n",
                gridLength, maxLinearThreads, gridLength * maxLinearThreads, width*height);


        struct pixel *inImageDevice;
        /*unsigned char *outImageDevice;*/
        uint16_t *outImageDevice;

        /* Allocate memory on the device for both images (in and out) */
        ret = hipMalloc((void **)&inImageDevice, width * height * sizeof(struct pixel));
        check_warn (ret == hipSuccess, "Failed to allocate memory for in image on the device");

        ret = hipMalloc((void **)&outImageDevice, width * height * 4 * sizeof(uint16_t));
        check_warn (ret == hipSuccess, "Failed to allocate memory for out image on the device");

        /* Copy The input image on the device */
        ret = hipMemcpy(inImageDevice, pInImage->data, width * height * sizeof(struct pixel),
                         hipMemcpyHostToDevice);
        check_warn (ret == hipSuccess, "Failed to copy the image to the device");


        /* Unnormalized version */
        uint16_t *pUnNormalizedOut = NULL;
        pUnNormalizedOut = (uint16_t *)calloc(width*height, sizeof(uint16_t));

        /* Allocate memory for the resulting image */
        pOutImage->width = pInImage->width;
        pOutImage->height = pInImage->height;
        pOutImage->type = RGBA;
        pOutImage->data = (unsigned char*)calloc(pOutImage->width * pOutImage->height, sizeof(struct pixel));
        /*check_mem(pOutImage->data);*/

        /* And launch the kernel */
        sobel_unnorm_kernel <<< nBlocks, threadsPerBlock >>> (inImageDevice, outImageDevice, width, height);

        ret = hipMemcpy(pUnNormalizedOut, outImageDevice,
                         width * height * sizeof(uint16_t), hipMemcpyDeviceToHost);
        check_warn (ret == hipSuccess, "Kernel failed: %s", hipGetErrorString(ret));


        /* Normalize, on CPU for the moment */
        uint16_t max = 0;
        uint32_t maxPos = 0;
        for (uint32_t i = 0; i < width*height; i++) {
                if (pUnNormalizedOut[i] > max) {
                        max = pUnNormalizedOut[i];
                        maxPos = i;
                }
        }
        printf("Max: %u, at %u\n", max, maxPos);

        for (uint32_t i = 0; i < width*height; i++) {
                unsigned char greyVal = (255 * pUnNormalizedOut[i]) / max;
                pOutImage->data[4*i] = greyVal;
                pOutImage->data[4*i + 1] = greyVal;
                pOutImage->data[4*i + 2] = greyVal;
                pOutImage->data[4*i + 3] = 255; /* full opacity */
        }



        hipFree(inImageDevice);
        hipFree(outImageDevice);
        return 0;
        //XXX need a cleanup for image in case of failure.

/*error:*/
        /*hipFree(inImageDevice);*/
        /*hipFree(outImageDevice);*/
        /*free_and_null(pOutImage->data);*/
        /*return -1;*/
}




} /* extern "C" */
